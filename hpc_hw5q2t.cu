#include "hip/hip_runtime.h"
#include <assert.h>
#include <numeric>
#include <random>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifdef PARALLEL
#include <omp.h>
#endif

constexpr auto N{256};
constexpr auto MaxNum{100};
constexpr auto TileSize{9};

template <typename T>
auto stencilCpu(T A[][N][N], const T B[][N][N], const std::size_t N) {
#ifdef PARALLEL
#pragma omp parallel for
#endif
  for (auto I{1}; I < N - 1; I++)
    for (auto J{1}; J < N - 1; J++)
      for (auto K{1}; K < N - 1; K++) {
        A[I][J][K] = 0.8 * (B[I - 1][J][K] + B[I + 1][J][K] + B[I][J - 1][K] +
                            B[I][J + 1][K] + B[I][J][K - 1] + B[I][J][K + 1]);
        // A[I][J][K] = I + J + K;
      }
}

template <typename T>
__global__ void stencilGpuTiled(T A[][N][N], const T B[][N][N],
                                const std::size_t N) {
  auto I{blockDim.x * blockIdx.x + threadIdx.x};
  auto J{blockDim.y * blockIdx.y + threadIdx.y};
  auto K{blockDim.z * blockIdx.z + threadIdx.z};

  auto X{threadIdx.x + 1};
  auto Y{threadIdx.y + 1};
  auto Z{threadIdx.z + 1};

  __shared__ T Tile[TileSize + 2][TileSize + 2][TileSize + 2];
  Tile[X][Y][Z] = B[I][J][K];

  if (I > 0 && X == 1) {
    Tile[X - 1][Y][Z] = B[I - 1][J][K];
  }
  if (I < N - 1 && X == TileSize) {
    Tile[X + 1][Y][Z] = B[I + 1][J][K];
  }
  if (J > 0 && Y == 1) {
    Tile[X][Y - 1][Z] = B[I][J - 1][K];
  }
  if (J < N - 1 && Y == TileSize) {
    Tile[X][Y + 1][Z] = B[I][J + 1][K];
  }
  if (K > 0 && Z == 1) {
    Tile[X][Y][Z - 1] = B[I][J][K - 1];
  }
  if (K < N - 1 && Z == TileSize) {
    Tile[X][Y][Z + 1] = B[I][J][K + 1];
  }

  __syncthreads();

  if ((I > N - 2) || (J > N - 2) || (K > N - 2) || (I < 1) || (J < 1) ||
      (K < 1))
    return;
  A[I][J][K] =
      0.8 * (Tile[X - 1][Y][Z] + Tile[X + 1][Y][Z] + Tile[X][Y - 1][Z] +
             Tile[X][Y + 1][Z] + Tile[X][Y][Z - 1] + Tile[X][Y][Z + 1]);
}

int main(int argc, char **argv) {

  auto Err{hipSuccess};

  // This will pick the best possible CUDA capable device
  int DevId;
  DevId = findCudaDevice(argc, (const char **)argv);

  // Get GPU information
  hipDeviceProp_t Props;
  checkCudaErrors(hipGetDevice(&DevId));
  checkCudaErrors(hipGetDeviceProperties(&Props, DevId));
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", DevId, Props.name,
         Props.major, Props.minor);

  using T = float;
  typedef T AT[N][N];
  auto NumberSize{N * N * N * sizeof(T)};

  // Populate matrix
  AT *HostA = nullptr;
  AT *HostB = nullptr;
  HostA = (AT *)malloc(NumberSize);
  HostB = (AT *)malloc(NumberSize);
  for (auto I{0}; I < N; ++I)
    for (auto J{0}; J < N; ++J)
      for (auto K{0}; K < N; ++K) {
        // HostB[I][J][K] = 1;
        HostB[I][J][K] = static_cast<T>(rand() % MaxNum);
        HostA[I][J][K] = 0;
      }

  float GpuElapsedTimeMs[4] = {0.0f, 0.0f, 0.0f, 0.0f};

  hipEvent_t Start, Stop;
  hipEventCreate(&Start);
  hipEventCreate(&Stop);
  AT *DeviceA;
  AT *DeviceB;

  // Allocate memory
  hipEventRecord(Start, 0);
  Err = hipMalloc(reinterpret_cast<void **>(&DeviceA), NumberSize);
  Err = hipMemset(DeviceA, static_cast<T> (0.0), NumberSize);
  Err = hipMalloc(reinterpret_cast<void **>(&DeviceB), NumberSize);
  Err = hipMemset(DeviceB, static_cast<T> (0.0), NumberSize);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[0], Start, Stop);
  if (Err != hipSuccess)
    std::cout << hipGetErrorString(Err) << std::endl;
  checkCudaErrors(Err);

  // Copy to device
  hipEventRecord(Start, 0);
  Err = hipMemcpy(DeviceB, HostB, NumberSize, hipMemcpyHostToDevice);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[1], Start, Stop);
  if (Err != hipSuccess)
    std::cout << hipGetErrorName(Err) << std::endl;
  checkCudaErrors(Err);

  // Compute
  //   const dim3 BlockSize(32, 8, 2);
  //   const dim3 GridSize((N + 31) / 32, (N + 7) / 8, (N + 1) / 2);
  const dim3 BlockSize(TileSize, TileSize, TileSize);
  const dim3 GridSize((N + TileSize - 1) / TileSize,
                      (N + TileSize - 1) / TileSize,
                      (N + TileSize - 1) / TileSize);
  hipEventRecord(Start, 0);
  stencilGpuTiled<<<GridSize, BlockSize>>>(DeviceA, DeviceB, N);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[2], Start, Stop);

  // Copy back
  AT *FromDeviceA = nullptr;
  FromDeviceA = (AT *)malloc(NumberSize);
  hipEventRecord(Start, 0);
  Err = hipMemcpy(FromDeviceA, DeviceA, NumberSize, hipMemcpyDeviceToHost);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[3], Start, Stop);
  if (Err != hipSuccess)
    std::cout << hipGetErrorString(Err) << std::endl;
  checkCudaErrors(Err);

  // CPU
  auto CpuElapsedTimeMs{0.0f};
  hipEventRecord(Start, 0);
  stencilCpu(HostA, HostB, N);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&CpuElapsedTimeMs, Start, Stop);

  auto ErrFlag{false};
  for (auto I{1}; I < N - 1; ++I)
    for (auto J{1}; J < N - 1; ++J)
      for (auto K{1}; K < N - 1; ++K)
        if (FromDeviceA[I][J][K] != HostA[I][J][K]) {
          ErrFlag = true;
          break;
        }
  std::cout << "Check passed? " << std::boolalpha << !ErrFlag << std::endl;
  std::cout << "GPU Time [ms]: ";
  std::for_each(std::begin(GpuElapsedTimeMs), std::end(GpuElapsedTimeMs),
                [&](float I) { std::cout << I << " "; });
  std::cout << std::endl;
  std::cout << "CPU Time [ms]: " << CpuElapsedTimeMs << std::endl;
  std::cout << "Speed up [xN]: "
            << CpuElapsedTimeMs /
                   (std::accumulate(std::begin(GpuElapsedTimeMs),
                                    std::end(GpuElapsedTimeMs), 0.0f))
            << " " << CpuElapsedTimeMs / GpuElapsedTimeMs[2] << std::endl;

  hipFree(DeviceA);
  hipFree(DeviceB);
  free(FromDeviceA);
  free(HostA);
  free(HostB);

  return EXIT_SUCCESS;
}
