#include "hip/hip_runtime.h"
#include <assert.h>
#include <numeric>
#include <random>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifdef PARALLEL
#include <omp.h>
#endif

constexpr auto N{256};
constexpr auto MaxNum{100};

template <typename T>
auto stencilCpu(T A[][N][N], const T B[][N][N], const std::size_t N) {
#ifdef PARALLEL
#pragma omp parallel for
#endif
  for (auto I{1}; I < N - 1; I++)
    for (auto J{1}; J < N - 1; J++)
      for (auto K{1}; K < N - 1; K++) {
        A[I][J][K] = 0.8 * (B[I - 1][J][K] + B[I + 1][J][K] + B[I][J - 1][K] +
                            B[I][J + 1][K] + B[I][J][K - 1] + B[I][J][K + 1]);
        // A[I][J][K] = I + J + K;
      }
}

// template <typename T> __global__ void stencilGpuTiled(T *A, const T *B) {
//   __shared__ T sA[TailSize][TailSize];
//   __shared__ T sB[TailSize][TailSize];

//   auto Row{blockDim.y * blockIdx.y + threadIdx.y};
//   auto Col{blockDim.x * blockIdx.x + threadIdx.x};
// }

template <typename T>
__global__ void stencilGpuNonTiled(T A[][N][N], const T B[][N][N],
                                   const std::size_t N) {
  auto I{blockDim.x * blockIdx.x + threadIdx.x};
  auto J{blockDim.y * blockIdx.y + threadIdx.y};
  auto K{blockDim.z * blockIdx.z + threadIdx.z};

  if ((I > N - 2) || (J > N - 2) || (K > N - 2) || (I < 1) || (J < 1) ||
      (K < 1))
    return;

  A[I][J][K] = 0.8 * (B[I - 1][J][K] + B[I + 1][J][K] + B[I][J - 1][K] +
                      B[I][J + 1][K] + B[I][J][K - 1] + B[I][J][K + 1]);
}

int main(int argc, char **argv) {

  auto Err{hipSuccess};

  // This will pick the best possible CUDA capable device
  int DevId;
  DevId = findCudaDevice(argc, (const char **)argv);

  // Get GPU information
  hipDeviceProp_t Props;
  checkCudaErrors(hipGetDevice(&DevId));
  checkCudaErrors(hipGetDeviceProperties(&Props, DevId));
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", DevId, Props.name,
         Props.major, Props.minor);

  using T = float;
  typedef T AT[N][N];
  auto NumberSize{N * N * N * sizeof(T)};

  // Populate matrix
  AT *HostA = nullptr;
  AT *HostB = nullptr;
  HostA = (AT *)malloc(NumberSize);
  HostB = (AT *)malloc(NumberSize);
  for (auto I{0}; I < N; ++I)
    for (auto J{0}; J < N; ++J)
      for (auto K{0}; K < N; ++K) {
        // HostB[I][J][K] = 1;
        HostB[I][J][K] = static_cast<T>(rand() % MaxNum);
        HostA[I][J][K] = 0;
      }

  float GpuElapsedTimeMs[4] = {0.0f, 0.0f, 0.0f, 0.0f};

  hipEvent_t Start, Stop;
  hipEventCreate(&Start);
  hipEventCreate(&Stop);
  AT *DeviceA;
  AT *DeviceB;

  // Allocate memory
  hipEventRecord(Start, 0);
  Err = hipMalloc(reinterpret_cast<void **>(&DeviceA), NumberSize);
  Err = hipMemset(DeviceA, 0.0f, NumberSize);
  Err = hipMalloc(reinterpret_cast<void **>(&DeviceB), NumberSize);
  Err = hipMemset(DeviceB, 0.0f, NumberSize);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[0], Start, Stop);
  if (Err != hipSuccess)
    std::cout << hipGetErrorString(Err) << std::endl;
  checkCudaErrors(Err);

  // Copy to device
  hipEventRecord(Start, 0);
  Err = hipMemcpy(DeviceB, HostB, NumberSize, hipMemcpyHostToDevice);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[1], Start, Stop);
  if (Err != hipSuccess)
    std::cout << hipGetErrorName(Err) << std::endl;
  checkCudaErrors(Err);

  // Compute
  const dim3 BlockSize(9, 9, 9);
  const dim3 GridSize((N + 8) / 9, (N + 8) / 9, (N + 8) / 9);
  hipEventRecord(Start, 0);
  stencilGpuNonTiled<<<GridSize, BlockSize>>>(DeviceA, DeviceB, N);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[2], Start, Stop);

  // Copy back
  AT *FromDeviceA = nullptr;
  FromDeviceA = (AT *)malloc(NumberSize);
  hipEventRecord(Start, 0);
  Err = hipMemcpy(FromDeviceA, DeviceA, NumberSize, hipMemcpyDeviceToHost);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[3], Start, Stop);
  if (Err != hipSuccess)
    std::cout << hipGetErrorString(Err) << std::endl;
  checkCudaErrors(Err);

  // CPU
  auto CpuElapsedTimeMs{0.0f};
  hipEventRecord(Start, 0);
  stencilCpu(HostA, HostB, N);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&CpuElapsedTimeMs, Start, Stop);

  auto ErrFlag{false};
  for (auto I{0}; I < N; ++I)
    for (auto J{0}; J < N; ++J)
      for (auto K{0}; K < N; ++K)
        if (FromDeviceA[I][J][K] != HostA[I][J][K]) {
          ErrFlag = true;
          break;
        }
  std::cout << "Check passed? " << std::boolalpha << !ErrFlag << std::endl;
  std::cout << "GPU Time [ms]: ";
  std::for_each(std::begin(GpuElapsedTimeMs), std::end(GpuElapsedTimeMs),
                [&](float I) { std::cout << I << " "; });
  std::cout << std::endl;
  std::cout << "CPU Time [ms]: " << CpuElapsedTimeMs << std::endl;
  std::cout << "Speed up [xN]: "
            << CpuElapsedTimeMs /
                   (std::accumulate(std::begin(GpuElapsedTimeMs),
                                    std::end(GpuElapsedTimeMs), 0.0f))
            << " " << CpuElapsedTimeMs / GpuElapsedTimeMs[2] << std::endl;

  hipFree(DeviceA);
  hipFree(DeviceB);
  free(FromDeviceA);
  free(HostA);
  free(HostB);

  return EXIT_SUCCESS;
}
